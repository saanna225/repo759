
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ int factorial(int n) {
  int res = 1;
  for (int i = 2; i <= n; ++i)
    res *= i;
  return res;
}

__global__ void compute_factorials() {
  int tid = threadIdx.x; // Only 1 block, so no need for blockIdx
  int num = tid + 1;     // Number range: 1 to 8
  int fact = factorial(num);
  printf("%d!=%d\n", num, fact);
}

int main() {
  compute_factorials<<<1, 8>>>();
  hipDeviceSynchronize(); // Ensure all printf results are flushed
  return 0;
}
