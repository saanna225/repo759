#include <cstdio>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include "vscale.cuh"


int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array size>\n", argv[0]);
        return 1;
    }

  unsigned int n = atoi(argv[1]);

    // Host memory
    float *hA = new float[n];
    float *hB = new float[n];

    // Random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distA(-10.0f, 10.0f);
    std::uniform_real_distribution<float> distB(0.0f, 1.0f);

    for (int i = 0; i < n; ++i) {
        hA[i] = distA(gen);
        hB[i] = distB(gen);
    }

    // Device memory
    float *dA, *dB;
    hipMalloc(&dA, n * sizeof(float));
    hipMalloc(&dB, n * sizeof(float));

    // Copy to device
    hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Kernel config
    int blockSize = 16;
    int gridSize = (n + blockSize - 1) / blockSize;
    vscale<<<gridSize, blockSize>>>(dA, dB, n);

    // Timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // Copy result back
    hipMemcpy(hB, dB, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("%.3f\n", ms);         // Time in ms
  //  printf("%.2f\n", hB[0]);      // First element
//    printf("%.2f\n", hB[n - 1]);  // Last element

    // Cleanup
    delete[] hA;
    delete[] hB;
    hipFree(dA);
    hipFree(dB);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

