#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>

__global__ void compute(int* dA, int a) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int x = threadIdx.x;
    int y = blockIdx.x;

    dA[tid] = a * x + y;
}

int main() {
    const int N = 16;
    int *dA, hA[N];

    // Generate a random integer 'a' between 1 and 10 (inclusive)
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(1, 10);
    int a = dist(gen);

    // Print random a
    printf("Random a = %d\n", a);

    // Allocate memory on the device
    hipMalloc((void**)&dA, N * sizeof(int));

    // Launch kernel: 2 blocks of 8 threads = 16 total threads
    compute<<<2, 8>>>(dA, a);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(hA, dA, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result from the host
    for (int i = 0; i < N; ++i) {
        printf("%d ", hA[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(dA);

    return 0;
}

